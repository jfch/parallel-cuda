#include "hip/hip_runtime.h"
// Simple SUDOKU probram in CUDA
// cmpe297_hw3_easysudoku.cu


/* *
 HW3 by Jiongfeng Chen

Parallelized GPU version of Sudoku Puzzle Algorithm and Its OPTIMIZATION

OPTIMIZATION Thinking:		
	ADD possible value array to save sesults from the last run to reduce iteration times 
   	So it only compare the possible values found from last run, and largely reduce the i
   	teration times of the loop in the following kernel code. It is extremely useful when 
   	the IS_OPTION is complex and require much excution time.
   	    temp = IS_OPTION(row, col, k); //old
   		temp = IS_OPTION(row, col, value_avail[mIter]); //optimized
					
Test Case: 
 	input matrix:
 	const int input_sdk[9][9] 
 		   =  {{0, 7, 0, 0, 6, 5, 0, 8, 0},
			  {6, 0, 0, 0, 3, 0, 4, 0, 0},
			  {0, 2, 0, 0, 4, 0, 7, 0, 0},
			  {8, 6, 0, 0, 0, 2, 5, 7, 0},
			  {0, 0, 7, 4, 0, 6, 1, 0, 0},
			  {0, 5, 2, 3, 0, 0, 0, 6, 4},
			  {0, 0, 8, 0, 2, 0, 0, 3, 0},
			  {0, 0, 5, 0, 8, 0, 0, 0, 1},
			  {0, 4, 0, 7, 1, 0, 0, 5, 0}};
Output:
	  0    *7*    0   |   0    *6*   *5*  |   0    *8*    0   
	 *6*    0     0   |   0    *3*    0   |  *4*    0     0   
	  0    *2*    0   |   0    *4*    0   |  *7*    0     0   
	---------------------------------------------------------------------
	 *8*   *6*    0   |   0     0    *2*  |  *5*   *7*    0   
	  0     0    *7*  |  *4*    0    *6*  |  *1*    0     0   
	  0    *5*   *2*  |  *3*    0     0   |   0    *6*   *4*  
	---------------------------------------------------------------------
	  0     0    *8*  |   0    *2*    0   |   0    *3*    0   
	  0     0    *5*  |   0    *8*    0   |   0     0    *1*  
	  0    *4*    0   |  *7*   *1*    0   |   0    *5*    0   

	 *4*   *7*   *1*  |  *9*   *6*   *5*  |  *3*   *8*   *2*  
	 *6*   *8*   *9*  |  *2*   *3*   *7*  |  *4*   *1*   *5*  
	 *5*   *2*   *3*  |  *8*   *4*   *1*  |  *7*   *9*   *6*  
	---------------------------------------------------------------------
	 *8*   *6*   *4*  |  *1*   *9*   *2*  |  *5*   *7*   *3*  
	 *3*   *9*   *7*  |  *4*   *5*   *6*  |  *1*   *2*   *8*  
	 *1*   *5*   *2*  |  *3*   *7*   *8*  |  *9*   *6*   *4*  
	---------------------------------------------------------------------
	 *9*   *1*   *8*  |  *5*   *2*   *4*  |  *6*   *3*   *7*  
	 *7*   *3*   *5*  |  *6*   *8*   *9*  |  *2*   *4*   *1*  
	 *2*   *4*   *6*  |  *7*   *1*   *3*  |  *8*   *5*   *9*  

	Kernel Execution Time: 37960 cycles
	Total cycles: 37960 

	
run:nvcc -I/usr/local/cuda/include -I. -lineinfo -arch=sm_53 -g -c cmpe297_hw3_sudoku.cu -o cmpe297_hw3_sudoku.o

*/


#include<stdio.h>
#include<string.h>
#include <hip/hip_runtime.h>

const int big_2x[9][9] = {{1, 1, 1, 1, 1, 1, 1, 1, 1},
			  {1, 1, 1, 1, 1, 1, 1, 1, 1},
			  {1, 1, 1, 1, 1, 1, 1, 1, 1},
			  {1, 1, 1, 1, 1, 1, 1, 1, 1},
			  {1, 1, 1, 1, 1, 1, 1, 1, 1},
			  {1, 1, 1, 1, 1, 1, 1, 1, 1},
			  {1, 1, 1, 1, 1, 1, 1, 1, 1},
			  {1, 1, 1, 1, 1, 1, 1, 1, 1},
			  {1, 1, 1, 1, 1, 1, 1, 1, 1}};

// input 9x9 sudoku : 
// - 1~9 : valid values 
// - 0 : no value is decided
const int input_sdk[9][9] =  {{0, 7, 0, 0, 6, 5, 0, 8, 0},
			  {6, 0, 0, 0, 3, 0, 4, 0, 0},
			  {0, 2, 0, 0, 4, 0, 7, 0, 0},
			  {8, 6, 0, 0, 0, 2, 5, 7, 0},
			  {0, 0, 7, 4, 0, 6, 1, 0, 0},
			  {0, 5, 2, 3, 0, 0, 0, 6, 4},
			  {0, 0, 8, 0, 2, 0, 0, 3, 0},
			  {0, 0, 5, 0, 8, 0, 0, 0, 1},
			  {0, 4, 0, 7, 1, 0, 0, 5, 0}};
typedef struct {
	int val[9][9]; // values that each entry can get
	int num_options[9][9]; // number of values that each entry can get
	int not_in_cell[9][9];	// values not in each 3x3 cell
	int not_in_row[9][9];	// values not in each row
	int not_in_col[9][9];	// values not in each column
} stContext;
stContext context;

void initialize_all();
void print_all();

#define WIDTH	9

#define IS_OPTION(row, col, k) \
			((shared_not_in_row[row][k] == 1) && \
			(shared_not_in_col[col][k] == 1) && \
			(shared_not_in_cell[row/3+(col/3)*3][k] == 1))? 1 : 0;

__device__ int
memcmp(stContext *input, int width)
{
	for(int i = 0; i< width; i++)
		for(int j = 0; j< width; j++)
			if(input->num_options[i][j] != 1)
				return 1;
	return 0;
}

#define FINISHED()	(memcmp(context, WIDTH) == 0? 1: 0)

// rule: numbers should be unique in each sub-array, each row, and each column
__global__ void k_Sudoku(stContext *context, int *runtime)
{
    const unsigned int col = threadIdx.x;
    const unsigned int row = threadIdx.y;
    
    //share memory to improve performance
		__shared__ int shared_val[WIDTH][WIDTH];
		__shared__ int shared_num_options[WIDTH][WIDTH];
		__shared__ int shared_not_in_cell[WIDTH][WIDTH];
		__shared__ int shared_not_in_row[WIDTH][WIDTH];
		__shared__ int shared_not_in_col[WIDTH][WIDTH];
		
		
		shared_val[row][col] = context->val[row][col];
		shared_num_options[row][col] = context->num_options[row][col];
		shared_not_in_cell[row][col] = context->not_in_cell[row][col];
		shared_not_in_row[row][col] = context->not_in_row[row][col];
		shared_not_in_col[row][col] = context->not_in_col[row][col];
		__syncthreads();
		
	    		
	//CLOCK: Measure the performance AFTER OPTIMIZATION
	int start_time = clock64();
    
    //printf("col %d row %d threads\n", col, row);
	while(!FINISHED())
	{
		//printf("again col %d row %d threads\n", col, row);		

		if(shared_num_options[row][col] > 1)
		{
					// Find values that are not in the row, col, and the 
					// 3x3 cell that (row, col) is belonged to.			
					
					
					int value = 0, temp;
					shared_num_options[row][col] = 0;
					
					//OPTIMIZATION, ADD possible value to save 
					//results from the last run to reduce iteration times 
					int value_avail[9];
					for(int kIter = 0; kIter < 9; kIter++)
						 value_avail[kIter] = 0;
						 
					//BEFORE OPTIMIZATION
					/*for(int k = 0; k < 9; k++)
					{
						temp = IS_OPTION(row, col, k);
						if(temp == 1)
						{
							shared_num_options[row][col]++;
							value = k;

						}
					}
					*/					
					//AFTER OPTIMIZATION, ADD possible value to save 
					//results from the last run to reduce iteration times
					int firstRun=1;
					int nIter =0;
					if(firstRun==1){	
						for(int k = 0; k < 9; k++)
						{
							temp = IS_OPTION(row, col, k);
							if(temp == 1)
							{
								shared_num_options[row][col]++;
								value = k;
								value_avail[nIter++] =value;
							}
						}
						firstRun=0;
					}
					else{
						int mIter=0;
						while(value_avail[mIter]!=0)
						{	//OPTIMIZATION
							//only compare the possible values found from last run
							temp = IS_OPTION(row, col, value_avail[mIter]);
							if(temp == 1)
							{
								shared_num_options[row][col]++;
								value = value_avail[mIter];
							}
							mIter++;
						}	
						value_avail[mIter] = 0;					
					}
					// If the above loop found only one value, 
					// set the value to (row, col)
					if(shared_num_options[row][col] == 1)
					{
						shared_not_in_row[row][value] = 0;
						shared_not_in_col[col][value] = 0;
						shared_not_in_cell[(row)/3+((col)/3)*3][value] = 0;
						shared_val[row][col] = value+1;
					}
		}
		context->num_options[row][col] = shared_num_options[row][col];
		__syncthreads();
	}//end while, find all grids
   
   	context->val[row][col] = shared_val[row][col];
	context->num_options[row][col] = shared_num_options[row][col];
	context->not_in_cell[row][col] = shared_not_in_cell[row][col];
	context->not_in_row[row][col] = shared_not_in_row[row][col];
	context->not_in_col[row][col] = shared_not_in_col[row][col];
	__syncthreads();

	//CLOCK: Measure the performance AFTER OPTIMIZATION
	int stop_time = clock64();
	runtime[row*WIDTH+col] = (int)(stop_time - start_time);
		

}

int main(int argc, char **argv)
{
    hipError_t err;

    initialize_all();
    print_all();

    stContext *k_context; //device matrix
    
	//CLOCK: Measure the performance AFTER OPTIMIZATION
	int* runtime; 				/*Exection cycles*/
	int* d_runtime;
	int runtime_size = WIDTH*WIDTH*sizeof(int); 
    hipMalloc((void**)&d_runtime, runtime_size);	
	runtime = (int *) malloc(runtime_size);
	memset(runtime, 0, runtime_size);
	
	// TODO: Allocate matrix in GPU device memory 
	// Print the matrix size to be used, and compute its size
    int size = 5*WIDTH*WIDTH*sizeof(int);
    //printf("[MatrixMul of %d x %d elements]\n", WIDTH, WIDTH);
    err = hipMalloc((void**)&k_context, size); 
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device data (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    // TODO: Copy the input matrix to GPU
    err = hipMemcpy(k_context, &context, size, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy data from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Assign as many threads as the matrix size so that
    // each thread can deal with one entry of the matrix
    dim3 dimBlock(WIDTH, WIDTH, 1);
    dim3 dimGrid(1, 1, 1);

	
    // TODO: Call the kernel function
    k_Sudoku<<<dimGrid,dimBlock>>>(k_context, d_runtime);
    
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Kernel execution failed (error code %s)\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    hipDeviceSynchronize();
	
	// TODO: Copy the result matrix from the GPU device memory
	err = hipMemcpy(&context, k_context, size, hipMemcpyDeviceToHost); 
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy data from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    hipDeviceSynchronize();
	
	//CLOCK: Measure the performance AFTER OPTIMIZATION
	//Copy the execution times from the GPU memory to HOST Code
	hipMemcpy(runtime, d_runtime, runtime_size, hipMemcpyDeviceToHost);
    hipDeviceSynchronize(); 
    //unsigned long long  elapsed_time = 0;
    int elapsed_time = 0;
    for(int i = 0; i < WIDTH*WIDTH; i++)
        if(elapsed_time < runtime[i])
            elapsed_time = runtime[i];
	
    // Print the result
    print_all();

	//printf("Kernel Execution Time: %llu cycles\n", elapsed_time);
	printf("Kernel Execution Time: %d cycles\n", elapsed_time);
	printf("Total cycles: %d \n", elapsed_time);

	// Free host memory
	free(runtime);

    // Free the device memory
    err = hipFree(k_context);
    err = hipFree(d_runtime);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free gpu data (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
	
    getchar();
	
    return 0;
}

void initialize_all()
{
    int i, j;

    memcpy(context.not_in_cell,big_2x, sizeof(big_2x));	
    memcpy(context.not_in_row,big_2x, sizeof(big_2x));	
    memcpy(context.not_in_col,big_2x, sizeof(big_2x));	
		
    for(i=0; i<9; i++){
    	for(j=0; j<9; j++){
            if(input_sdk[i][j] == 0)
            {
                context.val[i][j] = 0;
                context.num_options[i][j]=9;
            }
            else
            {
                context.val[i][j] = input_sdk[i][j];
                context.num_options[i][j] = 1;
                context.not_in_cell[i/3+(j/3)*3][input_sdk[i][j]-1] = 0;
                context.not_in_col[j][input_sdk[i][j]-1] = 0;
                context.not_in_row[i][input_sdk[i][j]-1] = 0;
            }
        }
    }
}


void print_all()
{
    int i, j, k;

    for(i=0; i<9; i++){
        for(j=0; j<9; j++){
            if(context.val[i][j] == 0)
                fprintf(stdout, "  %1d   ", context.val[i][j]);  
            else
                fprintf(stdout, " *%1d*  ", context.val[i][j]);  
            if((j==2)||(j==5)){
                fprintf(stdout, "| ");	
            }
        }
        fprintf(stdout, "\n");	
        if((i==2)||(i==5)){
            for(k=0; k<69; k++){
                fprintf(stdout, "-");	
            }
            fprintf(stdout, "\n");	
        }
    }
    fprintf(stdout, "\n");
}

