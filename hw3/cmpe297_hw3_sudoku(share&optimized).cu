#include "hip/hip_runtime.h"
// Simple SUDOKU probram in CUDA
// cmpe297_hw3_easysudoku.cu

#include<stdio.h>
#include<string.h>
#include <hip/hip_runtime.h>

const int big_2x[9][9] = {{1, 1, 1, 1, 1, 1, 1, 1, 1},
			  {1, 1, 1, 1, 1, 1, 1, 1, 1},
			  {1, 1, 1, 1, 1, 1, 1, 1, 1},
			  {1, 1, 1, 1, 1, 1, 1, 1, 1},
			  {1, 1, 1, 1, 1, 1, 1, 1, 1},
			  {1, 1, 1, 1, 1, 1, 1, 1, 1},
			  {1, 1, 1, 1, 1, 1, 1, 1, 1},
			  {1, 1, 1, 1, 1, 1, 1, 1, 1},
			  {1, 1, 1, 1, 1, 1, 1, 1, 1}};

// input 9x9 sudoku : 
// - 1~9 : valid values 
// - 0 : no value is decided
const int input_sdk[9][9] =  {{0, 7, 0, 0, 6, 5, 0, 8, 0},
			  {6, 0, 0, 0, 3, 0, 4, 0, 0},
			  {0, 2, 0, 0, 4, 0, 7, 0, 0},
			  {8, 6, 0, 0, 0, 2, 5, 7, 0},
			  {0, 0, 7, 4, 0, 6, 1, 0, 0},
			  {0, 5, 2, 3, 0, 0, 0, 6, 4},
			  {0, 0, 8, 0, 2, 0, 0, 3, 0},
			  {0, 0, 5, 0, 8, 0, 0, 0, 1},
			  {0, 4, 0, 7, 1, 0, 0, 5, 0}};
typedef struct {
	int val[9][9]; // values that each entry can get
	int num_options[9][9]; // number of values that each entry can get
	int not_in_cell[9][9];	// values not in each 3x3 cell
	int not_in_row[9][9];	// values not in each row
	int not_in_col[9][9];	// values not in each column
} stContext;
stContext context;

void initialize_all();
void print_all();

#define WIDTH	9

#define IS_OPTION(row, col, k) \
			((shared_not_in_row[row][k] == 1) && \
			(shared_not_in_col[col][k] == 1) && \
			(shared_not_in_cell[row/3+(col/3)*3][k] == 1))? 1 : 0;

__device__ int
memcmp(stContext *input, int width)
{
	for(int i = 0; i< width; i++)
		for(int j = 0; j< width; j++)
			if(input->num_options[i][j] != 1)
				return 1;
	return 0;
}

#define FINISHED()	(memcmp(context, WIDTH) == 0? 1: 0)

// rule: numbers should be unique in each sub-array, each row, and each column
__global__ void k_Sudoku(stContext *context)
{
    const unsigned int col = threadIdx.x;
    const unsigned int row = threadIdx.y;
    
    //share memory to improve performance
		__shared__ int shared_val[WIDTH][WIDTH];
		__shared__ int shared_num_options[WIDTH][WIDTH];
		__shared__ int shared_not_in_cell[WIDTH][WIDTH];
		__shared__ int shared_not_in_row[WIDTH][WIDTH];
		__shared__ int shared_not_in_col[WIDTH][WIDTH];
		
		
		shared_val[row][col] = context->val[row][col];
		shared_num_options[row][col] = context->num_options[row][col];
		shared_not_in_cell[row][col] = context->not_in_cell[row][col];
		shared_not_in_row[row][col] = context->not_in_row[row][col];
		shared_not_in_col[row][col] = context->not_in_col[row][col];
		__syncthreads();
    
    printf("col %d row %d threads\n", col, row);
	while(!FINISHED())
	{
		printf("again col %d row %d threads\n", col, row);		

		if(shared_num_options[row][col] > 1)
		{
					// Find values that are not in the row, col, and the 
					// 3x3 cell that (row, col) is belonged to.			
					
					
					int value = 0, temp;
					shared_num_options[row][col] = 0;
					
					//OPTIMIZATION, ADD possible value to save 
					//results from the last run to reduce iteration times 
					int value_avail[9];
					for(int kIter = 0; kIter < 9; kIter++)
						 value_avail[kIter] = 0;
						 
					//BEFORE OPTIMIZATION
					/*for(int k = 0; k < 9; k++)
					{
						temp = IS_OPTION(row, col, k);
						if(temp == 1)
						{
							shared_num_options[row][col]++;
							value = k;

						}
					}
					*/					
					//AFTER OPTIMIZATION, ADD possible value to save 
					//results from the last run to reduce iteration times
					int firstRun=1;
					int nIter =0;
					if(firstRun==1){	
						for(int k = 0; k < 9; k++)
						{
							temp = IS_OPTION(row, col, k);
							if(temp == 1)
							{
								shared_num_options[row][col]++;
								value = k;
								value_avail[nIter++] =value;
							}
						}
						firstRun=0;
					}
					else{
						int mIter=0;
						while(value_avail[mIter]!=0)
						{	//OPTIMIZATION
							//only compare the possible values found from last run
							temp = IS_OPTION(row, col, value_avail[mIter]);
							if(temp == 1)
							{
								shared_num_options[row][col]++;
								value = value_avail[mIter];
							}
							mIter++;
						}	
						value_avail[mIter] = 0;					
					}
					// If the above loop found only one value, 
					// set the value to (row, col)
					if(shared_num_options[row][col] == 1)
					{
						shared_not_in_row[row][value] = 0;
						shared_not_in_col[col][value] = 0;
						shared_not_in_cell[(row)/3+((col)/3)*3][value] = 0;
						shared_val[row][col] = value+1;
					}
		}
		context->num_options[row][col] = shared_num_options[row][col];
		__syncthreads();
	}//end while, find all grids
   
   	context->val[row][col] = shared_val[row][col];
	context->num_options[row][col] = shared_num_options[row][col];
	context->not_in_cell[row][col] = shared_not_in_cell[row][col];
	context->not_in_row[row][col] = shared_not_in_row[row][col];
	context->not_in_col[row][col] = shared_not_in_col[row][col];
	__syncthreads();
		

}

int main(int argc, char **argv)
{
    hipError_t err;

    initialize_all();
    print_all();

    stContext *k_context; //device matrix
	
	// TODO: Allocate matrix in GPU device memory 
	// Print the matrix size to be used, and compute its size
    int size = 5*WIDTH*WIDTH*sizeof(int);
    //printf("[MatrixMul of %d x %d elements]\n", WIDTH, WIDTH);
    err = hipMalloc((void**)&k_context, size); 
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device data (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    // TODO: Copy the input matrix to GPU
    err = hipMemcpy(k_context, &context, size, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy data from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Assign as many threads as the matrix size so that
    // each thread can deal with one entry of the matrix
    dim3 dimBlock(WIDTH, WIDTH, 1);
    dim3 dimGrid(1, 1, 1);

	
    // TODO: Call the kernel function
    k_Sudoku<<<dimGrid,dimBlock>>>(k_context);
    
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Kernel execution failed (error code %s)\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    hipDeviceSynchronize();
	
	// TODO: Copy the result matrix from the GPU device memory
	err = hipMemcpy(&context, k_context, size, hipMemcpyDeviceToHost); 
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy data from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    hipDeviceSynchronize();
	
	
    // Print the result
    print_all();

    // Free the device memory
    err = hipFree(k_context);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free gpu data (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
	
    getchar();
	
    return 0;
}

void initialize_all()
{
    int i, j;

    memcpy(context.not_in_cell,big_2x, sizeof(big_2x));	
    memcpy(context.not_in_row,big_2x, sizeof(big_2x));	
    memcpy(context.not_in_col,big_2x, sizeof(big_2x));	
		
    for(i=0; i<9; i++){
    	for(j=0; j<9; j++){
            if(input_sdk[i][j] == 0)
            {
                context.val[i][j] = 0;
                context.num_options[i][j]=9;
            }
            else
            {
                context.val[i][j] = input_sdk[i][j];
                context.num_options[i][j] = 1;
                context.not_in_cell[i/3+(j/3)*3][input_sdk[i][j]-1] = 0;
                context.not_in_col[j][input_sdk[i][j]-1] = 0;
                context.not_in_row[i][input_sdk[i][j]-1] = 0;
            }
        }
    }
}


void print_all()
{
    int i, j, k;

    for(i=0; i<9; i++){
        for(j=0; j<9; j++){
            if(context.val[i][j] == 0)
                fprintf(stdout, "  %1d   ", context.val[i][j]);  
            else
                fprintf(stdout, " *%1d*  ", context.val[i][j]);  
            if((j==2)||(j==5)){
                fprintf(stdout, "| ");	
            }
        }
        fprintf(stdout, "\n");	
        if((i==2)||(i==5)){
            for(k=0; k<69; k++){
                fprintf(stdout, "-");	
            }
            fprintf(stdout, "\n");	
        }
    }
    fprintf(stdout, "\n");
}

